

#include <hip/hip_runtime.h>
#include <iostream>

__device__ int mandelbrot(double a, double b, int maxIteration) {
    double real = a;
    double imag = b;
    int iteration = 0;
    while (real * real + imag * imag <= 4.0 && iteration < maxIteration) {
        double tempReal = real * real - imag * imag + a;
        imag = 2 * real * imag + b;
        real = tempReal;
        ++iteration;
    }
    return iteration;
}


__global__ void mandelbrotKernel(int width, int height, double minX, double minY,
                                 double scaleX, double scaleY, int offsetX, int offsetY,
                                 int maxIteration, double* output) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        double a = minX + (x + offsetX) * scaleX;
        double b = minY + (y + offsetY) * scaleY;

        int iteration = mandelbrot(a, b, maxIteration);
        output[y * width + x] = iteration;
    }
}

extern "C"
double* CudaExec(int width, int height, double minX, double minY, double scaleX, double scaleY,
              int offsetX, int offsetY, int maxIteration){

    double* h_output = new double[width * height];
    double* d_output;

    hipMalloc(&d_output, width * height * sizeof(double));

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((width + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (height + threadsPerBlock.y - 1) / threadsPerBlock.y);
    mandelbrotKernel<<<blocksPerGrid, threadsPerBlock>>>(width,height, minX, minY, scaleX, scaleY,offsetX, offsetY, maxIteration, d_output);
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cout << "CUDA error: " << hipGetErrorString(err);
    }

    hipMemcpy( h_output, d_output, width * height * sizeof(double), hipMemcpyDeviceToHost);
    hipFree(d_output);


    return h_output;
}
